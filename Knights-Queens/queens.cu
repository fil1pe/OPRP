#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "board.h"

__global__ void attack_queens(char *board, int i, int j, int m, int n){
	int k = blockIdx.x * blockDim.x + threadIdx.x + 1;

	if (i+k < m)
		board[(i+k)*n + j] = QUEEN_ATTACK;
	if (i-k >= 0)
		board[(i-k)*n + j] = QUEEN_ATTACK;
	if (j+k < n)
		board[i*n + j+k] = QUEEN_ATTACK;
	if (j-k >= 0)
		board[i*n + j-k] = QUEEN_ATTACK;
	if (i+k < m && j+k < n)
		board[(i+k)*n + j+k] = QUEEN_ATTACK;
	if (i+k < m && j-k >= 0)
		board[(i+k)*n + j-k] = QUEEN_ATTACK;
	if (i-k >= 0 && j+k < n)
		board[(i-k)*n + j+k] = QUEEN_ATTACK;
	if (i-k >= 0 && j-k >= 0)
		board[(i-k)*n + j-k] = QUEEN_ATTACK;
}

__host__ int queens(int qui, int quj, chessboard *board) {
	for(int i=qui; i<board->lin; i++){
		for(int j=quj; j<board->col; j++)
			if(board->board[i][j] == NO_PIECE){
				char *board_dev;
				hipMalloc((void**) &board_dev, board->lin * board->col);
				hipMemcpy(board_dev, board->board[0], board->lin * board->col, hipMemcpyHostToDevice);
				dim3 dimBlock (1);dim3 dimThreads(board->lin); // MELHORAR
				attack_queens<<< dimBlock, dimThreads>>>(board_dev, i, j, board->lin, board->col);
				hipDeviceSynchronize();
				hipMemcpy(board->board[0], board_dev, board->lin * board->col, hipMemcpyDeviceToHost);
				board->board[i][j] = QUEEN;
			}else if(board->board[i][j] == QUEEN_ATTACK || board->board[i][j] == DONT_PLACE_QUEEN){
				board->board[i][j] = SKIP;
				queens(i, j, board);
			}
	}

	return 0;
}