#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "board.h"
#include "queens.h"

// Marks positions where no queen must be placed
// in order to ensure the knight is safe
__global__ void attack_knights(char *board, int i, int j, int m, int n){
	int k = blockIdx.x * blockDim.x + threadIdx.x + 1;

	if (i+k < m && board[(i+k)*n + j] == NO_PIECE)
		board[(i+k)*n + j] = DONT_PLACE_QUEEN;
	if (i-k >= 0 && board[(i-k)*n + j] == NO_PIECE)
		board[(i-k)*n + j] = DONT_PLACE_QUEEN;
	if (j+k < n && board[i*n + j+k] == NO_PIECE)
		board[i*n + j+k] = DONT_PLACE_QUEEN;
	if (j-k >= 0 && board[i*n + j-k] == NO_PIECE)
		board[i*n + j-k] = DONT_PLACE_QUEEN;
	if (i+k < m && j+k < n && board[(i+k)*n + j+k] == NO_PIECE)
		board[(i+k)*n + j+k] = DONT_PLACE_QUEEN;
	if (i+k < m && j-k >= 0 && board[(i+k)*n + j-k] == NO_PIECE)
		board[(i+k)*n + j-k] = DONT_PLACE_QUEEN;
	if (i-k >= 0 && j+k < n && board[(i-k)*n + j+k] == NO_PIECE)
		board[(i-k)*n + j+k] = DONT_PLACE_QUEEN;
	if (i-k >= 0 && j-k >= 0 && board[(i-k)*n + j-k] == NO_PIECE)
		board[(i-k)*n + j-k] = DONT_PLACE_QUEEN;
}

// Marks the attacking positions relative to the knight at (i, j)
__host__ void attack_knights_host(char *board, int i, int j, int m, int n){
	if (i+2 < m && j - 1 >= 0)
		board[(i+2)*n + j-1] = KNIGHT_ATTACK;
	if (i-2 >= 0 && j - 1 >= 0)
		board[(i-2)*n + j-1] = KNIGHT_ATTACK;
	if (i+2 < m && j+1 < n)
		board[(i+2)*n + j+1] = KNIGHT_ATTACK;
	if (i-2 >= 0 && j + 1 < n)
		board[(i-2)*n + j+1] = KNIGHT_ATTACK;
	if (i+1 < m && j +2 >= 0)
		board[(i+1)*n + j+2] = KNIGHT_ATTACK;
	if (i-1 >= 0 && j + 2 >= 0)
		board[(i-1)*n + j+2] = KNIGHT_ATTACK;
	if (i+1 < m && j - 2 >= 0)
		board[(i+1)*n + j-2] = KNIGHT_ATTACK;
	if (i-1 >= 0 && j - 2 >= 0)
		board[(i-1)*n + j-2] = KNIGHT_ATTACK;

	// Tells GPU to block queens
	char *board_dev;
	hipMalloc(&board_dev, m*n);
	hipMemcpy(board_dev, board, m*n, hipMemcpyHostToDevice);
	dim3 dimBlock (1);dim3 dimThreads(m); // MELHORAR
	attack_knights <<<dimBlock, dimThreads>>>(board_dev, i, j, m, n);
	hipDeviceSynchronize();
	hipMemcpy(board, board_dev, m*n, hipMemcpyDeviceToHost);
}

// Places k knights
__host__ void knights(int k, chessboard *board){
	// If no knight is to be placed,
	// places queens
	if(k == 0){
		queens(0, 0, board);
		return;
	}

	// Counter for the number of knights already placed
	int cont = 0;

	// Places knights in linear ordering
	for(int i=0; i<board->lin; i++)
		for(int j=0; j<board->col; j++)
			if(board->board[i][j] == NO_PIECE || board->board[i][j] == DONT_PLACE_QUEEN){
				attack_knights_host(board->board[0], i, j, board->lin, board->col);
                board->board[i][j] = KNIGHT;
				
				// If already placed k knights,
				// it's time to place queens
				if(++cont == k){
					knights(0, board);
					return;
				}
			}
}